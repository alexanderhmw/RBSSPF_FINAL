#include "hip/hip_runtime.h"
#include"rbsspf_share.cuh"

__global__
void kernelSetupRandomSeed(int *seed, thrust::random::minstd_rand *rng)
{
    GetThreadID_1D(rid);
    if(rid>=RNGNUM) return;
    rng[rid]=thrust::minstd_rand(seed[rid]);
    return;
}

//====================================================

__host__
int hostCollectBeamCount(int *d_beamcount, int *h_beamcount, int tmppnum)
{
    hipMemcpy(h_beamcount,d_beamcount,sizeof(int)*tmppnum,hipMemcpyDeviceToHost);
    for(int i=1;i<tmppnum;i++)
    {
        h_beamcount[i]+=h_beamcount[i-1];
    }
    hipMemcpy(d_beamcount,h_beamcount,sizeof(int)*tmppnum,hipMemcpyHostToDevice);
    return h_beamcount[tmppnum-1];
}

__global__
void kernelSetupBeamArray(int *beamcount, int tmppnum, TrackerBeamEvaluator *beamevaluators)
{
    GetThreadID_1D(tmppid);
    if(tmppid>=tmppnum) return;
    int startid=tmppid>0?beamcount[tmppid-1]:0;
    int endid=beamcount[tmppid];
    for(int i=startid;i<endid;i++)
    {
        beamevaluators[i].tmppid=tmppid;
        beamevaluators[i].beamdelta=i-startid;
        beamevaluators[i].weight=0;
        beamevaluators[i].validflag=0;
    }
}

__global__
void kernelMeasureScan(TrackerBeamEvaluator *beamevaluators, int beamcount, TrackerParticle *tmpparticles, TrackerSampleControl *controls, double *scan, int beamnum, bool motionflag)
{
    GetThreadID_1D(measureid);
    if(measureid>=beamcount) return;
    TrackerBeamEvaluator evaluator=beamevaluators[measureid];

    int tmppid=evaluator.tmppid;
    TrackerParticle particle=tmpparticles[tmppid];

    int cid=particle.controlid;
    double iteration=motionflag?controls[cid].motioniteration:controls[cid].geometryiteration;
    if(iteration<1) return;
    double anneal=motionflag?controls[cid].motionanneal:controls[cid].geometryanneal;

    int beamid=particle.geometry.startbeamid+evaluator.beamdelta;
    int edgeid=beamid<particle.geometry.midbeamid?particle.geometry.startid:particle.geometry.midid;
    beamid%=beamnum;

    double bear=2*PI/beamnum*beamid-PI;
    double length=scan[beamid];

    double lx=cos(bear);
    double ly=sin(bear);

    double sa=particle.geometry.sa[edgeid];
    double sb=lx*particle.geometry.dy[edgeid]-particle.geometry.dx[edgeid]*ly;
    double l=sa/sb*particle.geometry.cn[edgeid];

    int nextedgeid=(edgeid+1)%4;
    double cn=lx*particle.geometry.dx[nextedgeid]+ly*particle.geometry.dy[nextedgeid];

    double l0=l-MARGIN0/cn;
    double l1=l-MARGIN1/cn;
    double l2=l;
    double l3=l+MARGIN2/cn;

    double delta,w1,w2;
    double tmplogweight;
    if(l<=NEARESTRING)
    {
        tmplogweight=0;
        beamevaluators[measureid].validflag=0;
    }
    else if(length<=l0)
    {
        delta=length-l0;
        w1=WEIGHT0-WEIGHT0;
        w2=WEIGHT1-WEIGHT0;
        tmplogweight=(w1+(w2-w1)*exp(-delta*delta/SIGMA));
        beamevaluators[measureid].validflag=0;
    }
    else if(length<=l1)
    {
        delta=length-l1;
        w1=WEIGHT1-WEIGHT0;
        w2=WEIGHT2-WEIGHT0;
        tmplogweight=(w1+(w2-w1)*exp(-delta*delta/SIGMA));
        beamevaluators[measureid].validflag=0;
    }
    else if(length<=l3)
    {
        delta=length-l2;
        w1=WEIGHT2-WEIGHT0;
        w2=2*w1;
        tmplogweight=(w1+(w2-w1)*exp(-delta*delta/SIGMA));
        beamevaluators[measureid].validflag=1;
    }
    else
    {
        delta=length-l3;
        w1=WEIGHT3-WEIGHT0;
        w2=WEIGHT2-WEIGHT0;
        tmplogweight=(w1+(w2-w1)*exp(-delta*delta/SIGMA));
        beamevaluators[measureid].validflag=0;
    }
    beamevaluators[measureid].weight=tmplogweight/anneal;
}

__global__
void kernelAccumulateWeight(double * weights, int * controlids, TrackerParticle * tmpparticles, int *beamcount, int tmppnum, TrackerBeamEvaluator *beamevaluators, TrackerParticle * tmpparticles_forward)
{
    GetThreadID_1D(tmppid);
    if(tmppid>=tmppnum) return;

    controlids[tmppid]=tmpparticles[tmppid].controlid;
    tmpparticles[tmppid].beamcount=0;
    weights[tmppid]=tmpparticles[tmppid].geometry.validflag?0:-100;

    int startid=tmppid>0?beamcount[tmppid-1]:0;
    int endid=beamcount[tmppid];
    for(int i=startid;i<endid;i++)
    {
        weights[tmppid]+=beamevaluators[i].weight;
        tmpparticles[tmppid].beamcount+=beamevaluators[i].validflag?1:0;
    }
    if(tmpparticles_forward!=NULL) tmpparticles_forward[tmppid].beamcount=tmpparticles[tmppid].beamcount;
}

//====================================================

__host__
void hostDownSampleIDs(int & startid, std::vector<int> & controlids, std::vector<double> & weights, int tmppnum, std::vector<TrackerSampleControl> & controls, int & pnum, std::vector<int> & sampleids, std::vector<int> & wcount, bool motionflag)
{
    int cid=controlids[startid];

    double maxlogweight=weights[startid];
    double minlogweight=weights[startid];
    int endid=startid;
    while(++endid<tmppnum)
    {
        if(cid!=controlids[endid]) break;
        maxlogweight=maxlogweight>weights[endid]?maxlogweight:weights[endid];
        minlogweight=minlogweight<weights[endid]?minlogweight:weights[endid];
    }

    double iteration=motionflag?controls[cid].motioniteration:controls[cid].geometryiteration;

    if(iteration<1)
    {
        int rqpn=(endid-startid)/SPN;
        for(int i=0;i<rqpn;i++)
        {
            sampleids[pnum+i]=startid+i*SPN;
            wcount[pnum+i]=0;
        }
        controls[cid].pnum=rqpn;
    }
    else
    {
        double maxscale=maxlogweight<30?1:30/maxlogweight;
        double minscale=minlogweight>-30?1:-30/minlogweight;
        double scale=maxscale<minscale?maxscale:minscale;

        weights[startid]=exp(weights[startid]*scale);
        for(int i=startid+1;i<endid;i++)
        {
            weights[i]=exp(weights[i]*scale);
            weights[i]+=weights[i-1];
        }

        int rqpn=endid-startid;
        if(motionflag)
        {
            rqpn=rqpn<MRQPN?rqpn:MRQPN;
        }
        else
        {
            rqpn=rqpn<GRQPN?rqpn:GRQPN;
        }

        double step=1.0/rqpn;
        int accuracy=1e6;
        double samplebase=(rand()%accuracy)*step/accuracy;
        double weightsum=weights[endid-1];

        controls[cid].pnum=0;
        for(int i=0,j=startid;i<rqpn;i++)
        {
            double sample=samplebase+i*step;
            while(j<endid)
            {
                if(sample>weights[j]/weightsum)
                {
                    j++;
                    continue;
                }
                else
                {
                    if(controls[cid].pnum==0||j!=sampleids[pnum+controls[cid].pnum-1])
                    {
                        sampleids[pnum+controls[cid].pnum]=j;
                        wcount[pnum+controls[cid].pnum]=1;
                        controls[cid].pnum++;
                    }
                    else
                    {
                        wcount[pnum+controls[cid].pnum-1]++;
                    }
                    break;
                }
            }
        }
    }
    startid=endid;
    pnum+=controls[cid].pnum;
}

__global__
void kernelDownSample(TrackerParticle * particles, int * sampleids, int * wcount, int pnum, TrackerParticle * tmpparticles)
{
    GetThreadID_1D(pid);
    if(pid>=pnum) return;

    particles[pid]=tmpparticles[sampleids[pid]];
    particles[pid].weight=wcount[pid]>0?wcount[pid]:tmpparticles[sampleids[pid]].weight;
}

//====================================================

__host__ __device__
void deviceBuildModel(TrackerParticle & particle, int beamnum)
{
    double c=cos(particle.state.theta);
    double s=sin(particle.state.theta);

    particle.geometry.cx[0]=c*particle.state.lf-s*particle.state.wl+particle.state.x; particle.geometry.cy[0]=s*particle.state.lf+c*particle.state.wl+particle.state.y;
    particle.geometry.cx[1]=c*particle.state.lf+s*particle.state.wr+particle.state.x; particle.geometry.cy[1]=s*particle.state.lf-c*particle.state.wr+particle.state.y;
    particle.geometry.cx[2]=-c*particle.state.lb+s*particle.state.wr+particle.state.x; particle.geometry.cy[2]=-s*particle.state.lb-c*particle.state.wr+particle.state.y;
    particle.geometry.cx[3]=-c*particle.state.lb-s*particle.state.wl+particle.state.x; particle.geometry.cy[3]=-s*particle.state.lb+c*particle.state.wl+particle.state.y;

    double width=particle.state.wl+particle.state.wr;
    double length=particle.state.lf+particle.state.lb;
    particle.geometry.dx[0]=(particle.geometry.cx[1]-particle.geometry.cx[0])/width; particle.geometry.dy[0]=(particle.geometry.cy[1]-particle.geometry.cy[0])/width;
    particle.geometry.dx[1]=(particle.geometry.cx[2]-particle.geometry.cx[1])/length; particle.geometry.dy[1]=(particle.geometry.cy[2]-particle.geometry.cy[1])/length;
    particle.geometry.dx[2]=(particle.geometry.cx[3]-particle.geometry.cx[2])/width; particle.geometry.dy[2]=(particle.geometry.cy[3]-particle.geometry.cy[2])/width;
    particle.geometry.dx[3]=(particle.geometry.cx[0]-particle.geometry.cx[3])/length; particle.geometry.dy[3]=(particle.geometry.cy[0]-particle.geometry.cy[3])/length;

    for(int i=0;i<4;i++)
    {
        particle.geometry.cn[i]=sqrt(particle.geometry.cx[i]*particle.geometry.cx[i]+particle.geometry.cy[i]*particle.geometry.cy[i]);
        particle.geometry.sa[i]=(particle.geometry.cx[i]*particle.geometry.dy[i]-particle.geometry.cy[i]*particle.geometry.dx[i])/particle.geometry.cn[i];
    }

    particle.geometry.validflag=0;
    double density=2*PI/beamnum;
    for(int i=0;i<4;i++)
    {
        int j=(i+1)%4;
        if(particle.geometry.sa[i]<=0&&particle.geometry.sa[j]>0)
        {
            particle.geometry.startid=(i+1)%4;
            double startbear=atan2(particle.geometry.cy[particle.geometry.startid],particle.geometry.cx[particle.geometry.startid])+PI;
            particle.geometry.startbeamid=int(startbear/density);

            particle.geometry.midid=(i+2)%4;
            double midbear=atan2(particle.geometry.cy[particle.geometry.midid],particle.geometry.cx[particle.geometry.midid])+PI;
            particle.geometry.midbeamid=int(midbear/density);

            particle.geometry.validflag=1;
        }
        else if(particle.geometry.sa[i]>0&&particle.geometry.sa[j]<=0)
        {
            particle.geometry.endid=(i+1)%4;
            double endbear=atan2(particle.geometry.cy[particle.geometry.endid],particle.geometry.cx[particle.geometry.endid])+PI;
            particle.geometry.endbeamid=int(endbear/density);

            particle.geometry.validflag=1;
        }
    }
    if(particle.geometry.validflag)
    {
        if(particle.geometry.midbeamid<particle.geometry.startbeamid)
        {
            particle.geometry.midbeamid+=beamnum;
        }
        if(particle.geometry.endbeamid<particle.geometry.startbeamid)
        {
            particle.geometry.endbeamid+=beamnum;
        }
        particle.geometry.beamcount=particle.geometry.endbeamid-particle.geometry.startbeamid+1;
    }
    else
    {
        particle.geometry.startid=-1;particle.geometry.startbeamid=-1;
        particle.geometry.midid=-1;particle.geometry.midbeamid=-1;
        particle.geometry.endid=-1;particle.geometry.endbeamid=-1;
        particle.geometry.beamcount=0;
    }
}

__host__
void hostBuildModel(Tracker & tracker, int beamnum)
{
    TrackerParticle particle;
    particle.state=tracker.mean;
    deviceBuildModel(particle,beamnum);

    tracker.cx[0]=particle.geometry.cx[0];tracker.cy[0]=particle.geometry.cy[0];
    tracker.cx[1]=particle.geometry.cx[1];tracker.cy[1]=particle.geometry.cy[1];
    tracker.cx[2]=particle.geometry.cx[2];tracker.cy[2]=particle.geometry.cy[2];
    tracker.cx[3]=particle.geometry.cx[3];tracker.cy[3]=particle.geometry.cy[3];

    tracker.startbeamid=particle.geometry.startbeamid;
    tracker.midbeamid=particle.geometry.midbeamid;
    tracker.endbeamid=particle.geometry.endbeamid;
}
